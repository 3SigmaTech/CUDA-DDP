#include "hip/hip_runtime.h"
﻿
#include "Header2.hpp"

using namespace CR2;


__global__ void CR2::addKernel(int *c, const int *a, const int *b)
{
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    hipDeviceSynchronize();
}
