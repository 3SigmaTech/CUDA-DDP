#include "hip/hip_runtime.h"
﻿
#include "Header3.hpp"

using namespace CR3;


__global__ void CR3::addKernel(int *c, const int *a, const int *b)
{
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    hipDeviceSynchronize();
}
